#include "hip/hip_runtime.h"
/*
 * This file belongs to the Galois project, a C++ library for exploiting parallelism.
 * The code is being released under the terms of the 3-Clause BSD License (a
 * copy is located in LICENSE.txt at the top-level directory).
 *
 * Copyright (C) 2018, The University of Texas at Austin. All rights reserved.
 * UNIVERSITY EXPRESSLY DISCLAIMS ANY AND ALL WARRANTIES CONCERNING THIS
 * SOFTWARE AND DOCUMENTATION, INCLUDING ANY WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR ANY PARTICULAR PURPOSE, NON-INFRINGEMENT AND WARRANTIES OF
 * PERFORMANCE, AND ANY WARRANTY THAT MIGHT OTHERWISE ARISE FROM COURSE OF
 * DEALING OR USAGE OF TRADE.  NO WARRANTY IS EITHER EXPRESS OR IMPLIED WITH
 * RESPECT TO THE USE OF THE SOFTWARE OR DOCUMENTATION. Under no circumstances
 * shall University be liable for incidental, special, indirect, direct or
 * consequential damages or loss of profits, interruption of business, or
 * related expenses which may arise from use of Software or Documentation,
 * including but not limited to those resulting from defects in Software and/or
 * Documentation, or loss or inaccuracy of data of any kind.
 */

/**
*
*   @edited by Jorge Silva <up201007483@alunos.dcc.fc.up.pt>
*/

/* -*- mode: c++ -*- */
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gg.h"
#include "ggcuda.h"
#include "hpr_wl_cuda.h"
#include "../hpr.h"
#include "worklist.h" // import work list

struct CUDA_Context {
    int device;
    int id;
    int pr_it;
    size_t nowned;
    size_t g_offset;
    CSRGraphTy hg;
    CSRGraphTy gg;
    Shared<float> pr[2];
    Shared<int> nout;
    //changed: added two worklists
    Worklist2 wl; //
    Worklist2 wl2;
};

__global__ void test_cuda_too(CSRGraphTy g) {
    int tid = TID_1D;
    int num_threads = TOTAL_THREADS_1D;

    for(int i = tid; i < g.nnodes; i+=num_threads) {
        printf("%d %d %d\n", tid, i, g.getOutDegree(i));
    }
}

__global__ void initialize_graph(CSRGraphTy g, index_type nowned, float *pcur, float *pnext, int *nout, Worklist2 wl) { //added work list
    int tid = TID_1D;
    int num_threads = TOTAL_THREADS_1D;
   
    for(int i = tid; i < nowned; i += num_threads) {
        wl.push(i); //changed: on the initialization add all nodes to the worklist
        pcur[i] = 1.0 - alpha;
        pnext[i] = 0;
  }
}


__global__ void initialize_nout(CSRGraphTy graph, index_type nowned, int *nout) {
    int tid = TID_1D;
    int num_threads = TOTAL_THREADS_1D;

    for(int i = tid; i < graph.nnodes; i += num_threads) {
        index_type edge_end = graph.getFirstEdge(i + 1);
        for(index_type e = graph.getFirstEdge(i); e < edge_end; e++) {
            index_type dst = graph.getAbsDestination(e);
            assert(dst < graph.nnodes);
            if(!(i >= nowned && dst >= nowned))
                atomicAdd(nout + dst, 1);
        }
    }
}

__global__ void pagerank(CSRGraphTy graph, int nowned, int *nout, float *pr, float *pr_next, Worklist2 inwl, Worklist2 outwl) {
    //printf ("CALLING PR %d\n", nowned);
    int tid = TID_1D;
    int num_threads = TOTAL_THREADS_1D;
    int i;

    for (int j = tid; j < inwl.nitems();  j+=num_threads) { // to read each element of the list
        inwl.pop_id(tid, i); //get the element from worklist
        float sum = 0.0;
        index_type edge_end = graph.getFirstEdge(i + 1);
        for(index_type e = graph.getFirstEdge(i); e < edge_end; e++) {
            index_type dst = graph.getAbsDestination(e);
            if(nout[dst] != 0)  // can nout == 0?
                sum += pr[dst] / nout[dst];      
            else 
                printf("WARNING: %d %d is zero\n", i, dst);
        }
        float value = (1.0 - alpha) * sum + alpha;
        float diff = fabs(value - pr[i]);
        pr_next[i] = value;
        //printf("PR FOR %d IS %f, %f, %f  ", i, value, pr[i], diff);
        if (diff > ERROR_THRESHOLD) { //if diff is bigger than tolerance then add the node for the next iteration
            outwl.push(i);
        }
    }
}

__global__ void test_graph(CSRGraphTy g, index_type nowned, float *pcur, float *pnext, int *nout, int id) {
    int tid = TID_1D;
    int num_threads = TOTAL_THREADS_1D;

    for(int i = tid; i < g.nnodes; i += num_threads) {
        printf("%d %d %d %d\n", id, tid, i, nout[i]); // i is LID!
    }
}


struct CUDA_Context *get_CUDA_context(int id) {
    struct CUDA_Context *p;
    p = (struct CUDA_Context *) calloc(1, sizeof(struct CUDA_Context));
    p->id = id;
    p->pr_it = 0;
    return p;
}

bool init_CUDA_context(struct CUDA_Context *ctx, int device) {
    struct hipDeviceProp_t dev;
    if(device == -1) {
        check_cuda(hipGetDevice(&device));
    } else {
    int count;
    check_cuda(hipGetDeviceCount(&count));
    if(device > count) {
        fprintf(stderr, "Error: Out-of-range GPU %d specified (%d total GPUs)", device, count);
        return false;
    }
    check_cuda(hipSetDevice(device));
    }
  
    ctx->device = device;
    check_cuda(hipGetDeviceProperties(&dev, device));
    fprintf(stderr, "%d: Using GPU %d: %s\n", ctx->id, device, dev.name);
    return true;
}

float getNodeValue_CUDA(struct CUDA_Context *ctx, unsigned LID) {
    float *pr = ctx->pr[ctx->pr_it].cpu_rd_ptr();
    return pr[LID];
}

void setNodeValue_CUDA(struct CUDA_Context *ctx, unsigned LID, float v) {
    float *pr = ctx->pr[ctx->pr_it].cpu_wr_ptr();
    pr[LID] = v;
}

void setNodeAttr_CUDA(struct CUDA_Context *ctx, unsigned LID, unsigned nout) {
    int *pnout = ctx->nout.cpu_wr_ptr();
    assert(LID >= ctx->nowned);
  
//  printf("setting %d %d %d\n", ctx->id, LID, nout);
    pnout[LID] = nout;
}

void setNodeAttr2_CUDA(struct CUDA_Context *ctx, unsigned LID, unsigned nout) {
    int *pnout = ctx->nout.cpu_wr_ptr();
    //printf("setting %d %d %d\n", ctx->id, LID, nout);

    assert(LID < ctx->nowned);
    pnout[LID] += nout;
}

unsigned getNodeAttr_CUDA(struct CUDA_Context *ctx, unsigned LID) {
    int *pnout = ctx->nout.cpu_rd_ptr();
    assert(LID < ctx->nowned);
    return pnout[LID];
}

unsigned getNodeAttr2_CUDA(struct CUDA_Context *ctx, unsigned LID) {
    int *pnout = ctx->nout.cpu_rd_ptr();
    assert(LID >= ctx->nowned);
    return pnout[LID];
}

void load_graph_CUDA(struct CUDA_Context *ctx, MarshalGraph &g) {
    CSRGraphTy &graph = ctx->hg;
    ctx->nowned = g.nowned;
    ctx->id = g.id;
    graph.nnodes = g.nnodes;
    graph.nedges = g.nedges;
    
    //initialize the lists
    ctx->wl = Worklist2(ctx->nowned);
    ctx->wl2 = Worklist2(ctx->nowned);

    if(!graph.allocOnHost()) {
        fprintf(stderr, "Unable to alloc space for graph!");
        exit(1);
    }
    memcpy(graph.row_start, g.row_start, sizeof(index_type) * (g.nnodes + 1));
    memcpy(graph.edge_dst, g.edge_dst, sizeof(index_type) * g.nedges);
    if(g.node_data)
        memcpy(graph.node_data, g.node_data, sizeof(node_data_type) * g.nnodes);
    if(g.edge_data)
        memcpy(graph.edge_data, g.edge_data, sizeof(edge_data_type) * g.nedges);

    graph.copy_to_gpu(ctx->gg);

    ctx->pr[0].alloc(graph.nnodes);
    ctx->pr[1].alloc(graph.nnodes);
    ctx->nout.alloc(graph.nnodes);

    printf("load_graph_GPU: %d owned nodes of total %d resident, %d edges\n", ctx->nowned, graph.nnodes, graph.nedges);  
}

void initialize_graph_cuda(struct CUDA_Context *ctx) {  
    ctx->nout.zero_gpu();
  
    initialize_graph<<<14, 256>>>(ctx->gg, ctx->nowned, ctx->pr[0].gpu_wr_ptr(), ctx->pr[1].gpu_wr_ptr(), ctx->nout.gpu_wr_ptr(), ctx->wl);

    initialize_nout<<<14, 256>>>(ctx->gg, ctx->nowned, ctx->nout.gpu_wr_ptr());
    check_cuda(hipDeviceSynchronize());
}

int pagerank_cuda(struct CUDA_Context *ctx) { 
    Worklist2 *inwl = &ctx->wl, *outwl = &ctx->wl2;
    pagerank<<<14, 256>>>(ctx->gg, ctx->nowned, ctx->nout.gpu_wr_ptr(), ctx->pr[ctx->pr_it].gpu_wr_ptr(), ctx->pr[ctx->pr_it ^ 1].gpu_wr_ptr(), *inwl, *outwl);
    ctx->pr_it ^= 1;  // not sure this is to be done here

    std::swap(ctx->wl, ctx->wl2); //switch lists
    ctx->wl2.reset();

    check_cuda(hipDeviceSynchronize());
    return ctx->wl.nitems();
}

void test_graph_cuda(struct CUDA_Context *ctx) {  
    test_graph<<<14, 256>>>(ctx->gg, ctx->nowned, ctx->pr[0].gpu_wr_ptr(), ctx->pr[1].gpu_wr_ptr(), ctx->nout.gpu_wr_ptr(), ctx->id);
    check_cuda(hipDeviceSynchronize());
}


void test_cuda(struct CUDA_Context *ctx) {
    printf("hello from cuda!\n");
    CSRGraphTy &gg = ctx->gg;

    test_cuda_too<<<1, 1>>>(gg);
    check_cuda(hipDeviceSynchronize());
}
