#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"
#include "hipcub/hipcub.hpp"
#include "cub/util_allocator.cuh"
#include "thread_work.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ tb_lb=True $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
struct ThreadWork t_work;
bool enable_lb = true;
#include "kernels/reduce.cuh"
#include "sssp_push_cuda.cuh"
static const int __tb_FirstItr_SSSP = TB_SIZE;
static const int __tb_SSSP = TB_SIZE;
__global__ void InitializeGraph(CSRGraph graph, unsigned int __begin, unsigned int __end, const uint32_t  local_infinity, unsigned long long local_src_node, uint32_t * p_dist_current, uint32_t * p_dist_old)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_dist_current[src] = (graph.node_data[src] == local_src_node) ? 0 : local_infinity;
      p_dist_old[src] = (graph.node_data[src] == local_src_node) ? 0 : local_infinity;
    }
  }
  // FP: "8 -> 9;
}
__global__ void FirstItr_SSSP_TB_LB(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_dist_current, uint32_t * p_dist_old, DynamicBitset& bitset_dist_current, int * thread_prefix_work_wl, unsigned int num_items, PipeContextT<Worklist2> thread_src_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ unsigned int total_work;
  __shared__ unsigned block_start_src_index;
  __shared__ unsigned block_end_src_index;
  unsigned my_work;
  unsigned src;
  unsigned int offset;
  unsigned int current_work;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  unsigned blockdim_x = BLOCK_DIM_X;
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  // FP: "9 -> 10;
  total_work = thread_prefix_work_wl[num_items - 1];
  // FP: "10 -> 11;
  my_work = ceilf((float)(total_work) / (float) nthreads);
  // FP: "11 -> 12;

  // FP: "12 -> 13;
  __syncthreads();
  // FP: "13 -> 14;

  // FP: "14 -> 15;
  if (my_work != 0)
  {
    current_work = tid;
  }
  // FP: "17 -> 18;
  for (unsigned i =0; i < my_work; i++)
  {
    unsigned int block_start_work;
    unsigned int block_end_work;
    if (threadIdx.x == 0)
    {
      if (current_work < total_work)
      {
        block_start_work = current_work;
        block_end_work=current_work + blockdim_x - 1;
        if (block_end_work >= total_work)
        {
          block_end_work = total_work - 1;
        }
        block_start_src_index = compute_src_and_offset(0, num_items - 1,  block_start_work+1, thread_prefix_work_wl, num_items,offset);
        block_end_src_index = compute_src_and_offset(0, num_items - 1, block_end_work+1, thread_prefix_work_wl, num_items, offset);
      }
    }
    __syncthreads();

    if (current_work < total_work)
    {
      unsigned src_index;
      index_type jj;
      src_index = compute_src_and_offset(block_start_src_index, block_end_src_index, current_work+1, thread_prefix_work_wl,num_items, offset);
      src= thread_src_wl.in_wl().dwl[src_index];
      jj = (graph).getFirstEdge(src)+ offset;
      {
        index_type dst;
        uint32_t new_dist;
        uint32_t old_dist;
        dst = graph.getAbsDestination(jj);
        new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
        old_dist = atomicTestMin(&p_dist_current[dst], new_dist);
        if (old_dist > new_dist)
        {
          bitset_dist_current.set(dst);
        }
      }
      current_work = current_work + nthreads;
    }
  }
  // FP: "48 -> 49;
}
__global__ void Inspect_FirstItr_SSSP(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_dist_current, uint32_t * p_dist_old, DynamicBitset& bitset_dist_current, PipeContextT<Worklist2> thread_work_wl, PipeContextT<Worklist2> thread_src_wl, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    int index;
    bool pop  = src < __end && ((( src < (graph).nnodes ) && ( (graph).getOutDegree(src) >= DEGREE_LIMIT)) ? true: false);
    if (pop)
    {
      p_dist_old[src]  = p_dist_current[src];
    }
    if (!pop)
    {
      continue;
    }
    if (pop)
    {
      index = thread_work_wl.in_wl().push_range(1) ;
      thread_src_wl.in_wl().push_range(1);
      thread_work_wl.in_wl().dwl[index] = (graph).getOutDegree(src);
      thread_src_wl.in_wl().dwl[index] = src;
    }
  }
  // FP: "14 -> 15;
}
__global__ void FirstItr_SSSP(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_dist_current, uint32_t * p_dist_old, DynamicBitset& bitset_dist_current, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_FirstItr_SSSP;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  unsigned d_limit = DEGREE_LIMIT;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end && ((( src < (graph).nnodes ) && ( (graph).getOutDegree(src) < DEGREE_LIMIT)) ? true: false);
    // FP: "7 -> 8;
    if (pop)
    {
      p_dist_old[src]  = p_dist_current[src];
    }
    // FP: "10 -> 11;
    // FP: "13 -> 14;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "14 -> 15;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "15 -> 16;
    _np_closure[threadIdx.x].src = src;
    // FP: "16 -> 17;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "19 -> 20;
    // FP: "20 -> 21;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "21 -> 22;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "22 -> 23;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "25 -> 26;
    __syncthreads();
    // FP: "26 -> 27;
    while (true)
    {
      // FP: "27 -> 28;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "30 -> 31;
      __syncthreads();
      // FP: "31 -> 32;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "32 -> 33;
        __syncthreads();
        // FP: "33 -> 34;
        break;
      }
      // FP: "35 -> 36;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "38 -> 39;
      __syncthreads();
      // FP: "39 -> 40;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "40 -> 41;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "43 -> 44;
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      // FP: "44 -> 45;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type jj;
        jj = ns +_np_j;
        {
          index_type dst;
          uint32_t new_dist;
          uint32_t old_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
          old_dist = atomicTestMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            bitset_dist_current.set(dst);
          }
        }
      }
      // FP: "57 -> 58;
      __syncthreads();
    }
    // FP: "59 -> 60;

    // FP: "60 -> 61;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "61 -> 62;
      const int _np_laneid = hipcub::LaneId();
      // FP: "62 -> 63;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type jj;
          jj = _np_w_start +_np_ii;
          {
            index_type dst;
            uint32_t new_dist;
            uint32_t old_dist;
            dst = graph.getAbsDestination(jj);
            new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
            old_dist = atomicTestMin(&p_dist_current[dst], new_dist);
            if (old_dist > new_dist)
            {
              bitset_dist_current.set(dst);
            }
          }
        }
      }
      // FP: "85 -> 86;
      __syncthreads();
      // FP: "86 -> 87;
    }

    // FP: "87 -> 88;
    __syncthreads();
    // FP: "88 -> 89;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "89 -> 90;
    while (_np.work())
    {
      // FP: "90 -> 91;
      int _np_i =0;
      // FP: "91 -> 92;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "92 -> 93;
      __syncthreads();
      // FP: "93 -> 94;

      // FP: "94 -> 95;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type jj;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        jj= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          uint32_t new_dist;
          uint32_t old_dist;
          dst = graph.getAbsDestination(jj);
          new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
          old_dist = atomicTestMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            bitset_dist_current.set(dst);
          }
        }
      }
      // FP: "108 -> 109;
      _np.execute_round_done(ITSIZE);
      // FP: "109 -> 110;
      __syncthreads();
    }
    // FP: "111 -> 112;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
  }
  // FP: "113 -> 114;
}
__global__ void SSSP_TB_LB(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t local_priority, uint32_t * p_dist_current, uint32_t * p_dist_old, DynamicBitset& bitset_dist_current, HGAccumulator<unsigned int> active_vertices, HGAccumulator<unsigned int> work_edges, int * thread_prefix_work_wl, unsigned int num_items, PipeContextT<Worklist2> thread_src_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ unsigned int total_work;
  __shared__ unsigned block_start_src_index;
  __shared__ unsigned block_end_src_index;
  unsigned my_work;
  unsigned src;
  unsigned int offset;
  unsigned int current_work;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  unsigned blockdim_x = BLOCK_DIM_X;
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  // FP: "9 -> 10;
  total_work = thread_prefix_work_wl[num_items - 1];
  // FP: "10 -> 11;
  my_work = ceilf((float)(total_work) / (float) nthreads);
  // FP: "11 -> 12;

  // FP: "12 -> 13;
  __syncthreads();
  // FP: "13 -> 14;

  // FP: "14 -> 15;
  if (my_work != 0)
  {
    current_work = tid;
  }
  // FP: "17 -> 18;
  for (unsigned i =0; i < my_work; i++)
  {
    unsigned int block_start_work;
    unsigned int block_end_work;
    if (threadIdx.x == 0)
    {
      if (current_work < total_work)
      {
        block_start_work = current_work;
        block_end_work=current_work + blockdim_x - 1;
        if (block_end_work >= total_work)
        {
          block_end_work = total_work - 1;
        }
        block_start_src_index = compute_src_and_offset(0, num_items - 1,  block_start_work+1, thread_prefix_work_wl, num_items,offset);
        block_end_src_index = compute_src_and_offset(0, num_items - 1, block_end_work+1, thread_prefix_work_wl, num_items, offset);
      }
    }
    __syncthreads();

    if (current_work < total_work)
    {
      unsigned src_index;
      index_type jj;
      src_index = compute_src_and_offset(block_start_src_index, block_end_src_index, current_work+1, thread_prefix_work_wl,num_items, offset);
      src= thread_src_wl.in_wl().dwl[src_index];
      jj = (graph).getFirstEdge(src)+ offset;
      {
        index_type dst;
        uint32_t new_dist;
        uint32_t old_dist;
        work_edges.reduce( 1);
        dst = graph.getAbsDestination(jj);
        new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
        old_dist = atomicTestMin(&p_dist_current[dst], new_dist);
        if (old_dist > new_dist)
        {
          bitset_dist_current.set(dst);
        }
      }
      current_work = current_work + nthreads;
    }
  }
  // FP: "49 -> 50;
}
__global__ void Inspect_SSSP(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t local_priority, uint32_t * p_dist_current, uint32_t * p_dist_old, DynamicBitset& bitset_dist_current, HGAccumulator<unsigned int> active_vertices, HGAccumulator<unsigned int> work_edges, PipeContextT<Worklist2> thread_work_wl, PipeContextT<Worklist2> thread_src_wl, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<unsigned int, TB_SIZE>::TempStorage active_vertices_ts;
  __shared__ hipcub::BlockReduce<unsigned int, TB_SIZE>::TempStorage work_edges_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  active_vertices.thread_entry();
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  work_edges.thread_entry();
  // FP: "5 -> 6;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    int index;
    bool pop  = src < __end && ((( src < (graph).nnodes ) && ( (graph).getOutDegree(src) >= DEGREE_LIMIT)) ? true: false);
    if (pop)
    {
      if (p_dist_old[src] > p_dist_current[src])
      {
        active_vertices.reduce( 1);
        if (local_priority > p_dist_current[src])
        {
          p_dist_old[src] = p_dist_current[src];
        }
        else
        {
          pop = false;
        }
      }
      else
      {
        pop = false;
      }
    }
    if (!pop)
    {
      continue;
    }
    if (pop)
    {
      index = thread_work_wl.in_wl().push_range(1) ;
      thread_src_wl.in_wl().push_range(1);
      thread_work_wl.in_wl().dwl[index] = (graph).getOutDegree(src);
      thread_src_wl.in_wl().dwl[index] = src;
    }
  }
  // FP: "25 -> 26;
  active_vertices.thread_exit<hipcub::BlockReduce<unsigned int, TB_SIZE> >(active_vertices_ts);
  // FP: "26 -> 27;
  work_edges.thread_exit<hipcub::BlockReduce<unsigned int, TB_SIZE> >(work_edges_ts);
  // FP: "27 -> 28;
}
__global__ void SSSP(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t local_priority, uint32_t * p_dist_current, uint32_t * p_dist_old, DynamicBitset& bitset_dist_current, HGAccumulator<unsigned int> active_vertices, HGAccumulator<unsigned int> work_edges, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_SSSP;
  __shared__ hipcub::BlockReduce<unsigned int, TB_SIZE>::TempStorage active_vertices_ts;
  __shared__ hipcub::BlockReduce<unsigned int, TB_SIZE>::TempStorage work_edges_ts;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  unsigned d_limit = DEGREE_LIMIT;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  active_vertices.thread_entry();
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  work_edges.thread_entry();
  // FP: "9 -> 10;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "10 -> 11;
    bool pop  = src < __end && ((( src < (graph).nnodes ) && ( (graph).getOutDegree(src) < DEGREE_LIMIT)) ? true: false);
    // FP: "11 -> 12;
    if (pop)
    {
      if (p_dist_old[src] > p_dist_current[src])
      {
        active_vertices.reduce( 1);
        if (local_priority > p_dist_current[src])
        {
          p_dist_old[src] = p_dist_current[src];
        }
        else
        {
          pop = false;
        }
      }
      else
      {
        pop = false;
      }
    }
    // FP: "19 -> 20;
    // FP: "22 -> 23;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "23 -> 24;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "24 -> 25;
    _np_closure[threadIdx.x].src = src;
    // FP: "25 -> 26;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "28 -> 29;
    // FP: "29 -> 30;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "30 -> 31;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "31 -> 32;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "34 -> 35;
    __syncthreads();
    // FP: "35 -> 36;
    while (true)
    {
      // FP: "36 -> 37;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "39 -> 40;
      __syncthreads();
      // FP: "40 -> 41;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "41 -> 42;
        __syncthreads();
        // FP: "42 -> 43;
        break;
      }
      // FP: "44 -> 45;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "47 -> 48;
      __syncthreads();
      // FP: "48 -> 49;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "49 -> 50;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "52 -> 53;
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      // FP: "53 -> 54;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type jj;
        jj = ns +_np_j;
        {
          index_type dst;
          uint32_t new_dist;
          uint32_t old_dist;
          work_edges.reduce( 1);
          dst = graph.getAbsDestination(jj);
          new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
          old_dist = atomicTestMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            bitset_dist_current.set(dst);
          }
        }
      }
      // FP: "67 -> 68;
      __syncthreads();
    }
    // FP: "69 -> 70;

    // FP: "70 -> 71;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "71 -> 72;
      const int _np_laneid = hipcub::LaneId();
      // FP: "72 -> 73;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type jj;
          jj = _np_w_start +_np_ii;
          {
            index_type dst;
            uint32_t new_dist;
            uint32_t old_dist;
            work_edges.reduce( 1);
            dst = graph.getAbsDestination(jj);
            new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
            old_dist = atomicTestMin(&p_dist_current[dst], new_dist);
            if (old_dist > new_dist)
            {
              bitset_dist_current.set(dst);
            }
          }
        }
      }
      // FP: "96 -> 97;
      __syncthreads();
      // FP: "97 -> 98;
    }

    // FP: "98 -> 99;
    __syncthreads();
    // FP: "99 -> 100;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "100 -> 101;
    while (_np.work())
    {
      // FP: "101 -> 102;
      int _np_i =0;
      // FP: "102 -> 103;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "103 -> 104;
      __syncthreads();
      // FP: "104 -> 105;

      // FP: "105 -> 106;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type jj;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        jj= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          uint32_t new_dist;
          uint32_t old_dist;
          work_edges.reduce( 1);
          dst = graph.getAbsDestination(jj);
          new_dist = graph.getAbsWeight(jj) + p_dist_current[src];
          old_dist = atomicTestMin(&p_dist_current[dst], new_dist);
          if (old_dist > new_dist)
          {
            bitset_dist_current.set(dst);
          }
        }
      }
      // FP: "120 -> 121;
      _np.execute_round_done(ITSIZE);
      // FP: "121 -> 122;
      __syncthreads();
    }
    // FP: "123 -> 124;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
  }
  // FP: "127 -> 128;
  active_vertices.thread_exit<hipcub::BlockReduce<unsigned int, TB_SIZE> >(active_vertices_ts);
  // FP: "128 -> 129;
  work_edges.thread_exit<hipcub::BlockReduce<unsigned int, TB_SIZE> >(work_edges_ts);
  // FP: "129 -> 130;
}
__global__ void SSSPSanityCheck(CSRGraph graph, unsigned int __begin, unsigned int __end, const uint32_t  local_infinity, uint32_t * p_dist_current, HGAccumulator<uint64_t> DGAccumulator_sum, HGAccumulator<uint64_t> dg_avg, HGReduceMax<uint32_t> DGMax)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<uint64_t, TB_SIZE>::TempStorage DGAccumulator_sum_ts;
  __shared__ hipcub::BlockReduce<uint64_t, TB_SIZE>::TempStorage dg_avg_ts;
  __shared__ hipcub::BlockReduce<uint32_t, TB_SIZE>::TempStorage DGMax_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  DGAccumulator_sum.thread_entry();
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  dg_avg.thread_entry();
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  DGMax.thread_entry();
  // FP: "7 -> 8;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      if (p_dist_current[src] < local_infinity)
      {
        DGAccumulator_sum.reduce( 1);
        DGMax.reduce(p_dist_current[src]);
        dg_avg.reduce( p_dist_current[src]);
      }
    }
  }
  // FP: "17 -> 18;
  DGAccumulator_sum.thread_exit<hipcub::BlockReduce<uint64_t, TB_SIZE> >(DGAccumulator_sum_ts);
  // FP: "18 -> 19;
  dg_avg.thread_exit<hipcub::BlockReduce<uint64_t, TB_SIZE> >(dg_avg_ts);
  // FP: "19 -> 20;
  DGMax.thread_exit<hipcub::BlockReduce<uint32_t, TB_SIZE> >(DGMax_ts);
  // FP: "20 -> 21;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, const uint32_t & local_infinity, unsigned long long local_src_node, struct CUDA_Context*  ctx)
{
  t_work.init_thread_work(ctx->gg.nnodes);
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  InitializeGraph <<<blocks, threads>>>(ctx->gg, __begin, __end, local_infinity, local_src_node, ctx->dist_current.data.gpu_wr_ptr(), ctx->dist_old.data.gpu_wr_ptr());
  hipDeviceSynchronize();
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_allNodes_cuda(const uint32_t & local_infinity, unsigned long long local_src_node, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->gg.nnodes, local_infinity, local_src_node, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_masterNodes_cuda(const uint32_t & local_infinity, unsigned long long local_src_node, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, local_infinity, local_src_node, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_nodesWithEdges_cuda(const uint32_t & local_infinity, unsigned long long local_src_node, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->numNodesWithEdges, local_infinity, local_src_node, ctx);
  // FP: "2 -> 3;
}
void FirstItr_SSSP_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  if (enable_lb)
  {
    t_work.reset_thread_work();
    Inspect_FirstItr_SSSP <<<blocks, __tb_FirstItr_SSSP>>>(ctx->gg, __begin, __end, ctx->dist_current.data.gpu_wr_ptr(), ctx->dist_old.data.gpu_wr_ptr(), *(ctx->dist_current.is_updated.gpu_rd_ptr()), t_work.thread_work_wl, t_work.thread_src_wl, enable_lb);
    hipDeviceSynchronize();
    int num_items = t_work.thread_work_wl.in_wl().nitems();
    if (num_items != 0)
    {
      t_work.compute_prefix_sum();
      hipDeviceSynchronize();
      FirstItr_SSSP_TB_LB <<<blocks, __tb_FirstItr_SSSP>>>(ctx->gg, __begin, __end, ctx->dist_current.data.gpu_wr_ptr(), ctx->dist_old.data.gpu_wr_ptr(), *(ctx->dist_current.is_updated.gpu_rd_ptr()), t_work.thread_prefix_work_wl.gpu_wr_ptr(), num_items, t_work.thread_src_wl);
      hipDeviceSynchronize();
    }
  }
  FirstItr_SSSP <<<blocks, __tb_FirstItr_SSSP>>>(ctx->gg, __begin, __end, ctx->dist_current.data.gpu_wr_ptr(), ctx->dist_old.data.gpu_wr_ptr(), *(ctx->dist_current.is_updated.gpu_rd_ptr()), enable_lb);
  hipDeviceSynchronize();
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void FirstItr_SSSP_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  FirstItr_SSSP_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void FirstItr_SSSP_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  FirstItr_SSSP_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void FirstItr_SSSP_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  FirstItr_SSSP_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void SSSP_cuda(unsigned int  __begin, unsigned int  __end, unsigned int & active_vertices, unsigned int & work_edges, uint32_t local_priority, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<unsigned int> _active_vertices;
  HGAccumulator<unsigned int> _work_edges;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<unsigned int> active_verticesval  = Shared<unsigned int>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(active_verticesval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _active_vertices.rv = active_verticesval.gpu_wr_ptr();
  // FP: "8 -> 9;
  Shared<unsigned int> work_edgesval  = Shared<unsigned int>(1);
  // FP: "9 -> 10;
  // FP: "10 -> 11;
  *(work_edgesval.cpu_wr_ptr()) = 0;
  // FP: "11 -> 12;
  _work_edges.rv = work_edgesval.gpu_wr_ptr();
  // FP: "12 -> 13;
  if (enable_lb)
  {
    t_work.reset_thread_work();
    Inspect_SSSP <<<blocks, __tb_SSSP>>>(ctx->gg, __begin, __end, local_priority, ctx->dist_current.data.gpu_wr_ptr(), ctx->dist_old.data.gpu_wr_ptr(), *(ctx->dist_current.is_updated.gpu_rd_ptr()), _active_vertices, _work_edges, t_work.thread_work_wl, t_work.thread_src_wl, enable_lb);
    hipDeviceSynchronize();
    int num_items = t_work.thread_work_wl.in_wl().nitems();
    if (num_items != 0)
    {
      t_work.compute_prefix_sum();
      hipDeviceSynchronize();
      SSSP_TB_LB <<<blocks, __tb_SSSP>>>(ctx->gg, __begin, __end, local_priority, ctx->dist_current.data.gpu_wr_ptr(), ctx->dist_old.data.gpu_wr_ptr(), *(ctx->dist_current.is_updated.gpu_rd_ptr()), _active_vertices, _work_edges, t_work.thread_prefix_work_wl.gpu_wr_ptr(), num_items, t_work.thread_src_wl);
      hipDeviceSynchronize();
    }
  }
  SSSP <<<blocks, __tb_SSSP>>>(ctx->gg, __begin, __end, local_priority, ctx->dist_current.data.gpu_wr_ptr(), ctx->dist_old.data.gpu_wr_ptr(), *(ctx->dist_current.is_updated.gpu_rd_ptr()), _active_vertices, _work_edges, enable_lb);
  hipDeviceSynchronize();
  // FP: "13 -> 14;
  check_cuda_kernel;
  // FP: "14 -> 15;
  active_vertices = *(active_verticesval.cpu_rd_ptr());
  // FP: "15 -> 16;
  work_edges = *(work_edgesval.cpu_rd_ptr());
  // FP: "16 -> 17;
}
void SSSP_allNodes_cuda(unsigned int & active_vertices, unsigned int & work_edges, uint32_t local_priority, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  SSSP_cuda(0, ctx->gg.nnodes, active_vertices, work_edges, local_priority, ctx);
  // FP: "2 -> 3;
}
void SSSP_masterNodes_cuda(unsigned int & active_vertices, unsigned int & work_edges, uint32_t local_priority, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  SSSP_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, active_vertices, work_edges, local_priority, ctx);
  // FP: "2 -> 3;
}
void SSSP_nodesWithEdges_cuda(unsigned int & active_vertices, unsigned int & work_edges, uint32_t local_priority, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  SSSP_cuda(0, ctx->numNodesWithEdges, active_vertices, work_edges, local_priority, ctx);
  // FP: "2 -> 3;
}
void SSSPSanityCheck_cuda(unsigned int  __begin, unsigned int  __end, uint64_t & DGAccumulator_sum, uint64_t & dg_avg, uint32_t & DGMax, const uint32_t & local_infinity, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<uint64_t> _DGAccumulator_sum;
  HGAccumulator<uint64_t> _dg_avg;
  HGReduceMax<uint32_t> _DGMax;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<uint64_t> DGAccumulator_sumval  = Shared<uint64_t>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(DGAccumulator_sumval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _DGAccumulator_sum.rv = DGAccumulator_sumval.gpu_wr_ptr();
  // FP: "8 -> 9;
  Shared<uint64_t> dg_avgval  = Shared<uint64_t>(1);
  // FP: "9 -> 10;
  // FP: "10 -> 11;
  *(dg_avgval.cpu_wr_ptr()) = 0;
  // FP: "11 -> 12;
  _dg_avg.rv = dg_avgval.gpu_wr_ptr();
  // FP: "12 -> 13;
  Shared<uint32_t> DGMaxval  = Shared<uint32_t>(1);
  // FP: "13 -> 14;
  // FP: "14 -> 15;
  *(DGMaxval.cpu_wr_ptr()) = 0;
  // FP: "15 -> 16;
  _DGMax.rv = DGMaxval.gpu_wr_ptr();
  // FP: "16 -> 17;
  SSSPSanityCheck <<<blocks, threads>>>(ctx->gg, __begin, __end, local_infinity, ctx->dist_current.data.gpu_wr_ptr(), _DGAccumulator_sum, _dg_avg, _DGMax);
  hipDeviceSynchronize();
  // FP: "17 -> 18;
  check_cuda_kernel;
  // FP: "18 -> 19;
  DGAccumulator_sum = *(DGAccumulator_sumval.cpu_rd_ptr());
  // FP: "19 -> 20;
  dg_avg = *(dg_avgval.cpu_rd_ptr());
  // FP: "20 -> 21;
  DGMax = *(DGMaxval.cpu_rd_ptr());
  // FP: "21 -> 22;
}
void SSSPSanityCheck_allNodes_cuda(uint64_t & DGAccumulator_sum, uint64_t & dg_avg, uint32_t & DGMax, const uint32_t & local_infinity, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  SSSPSanityCheck_cuda(0, ctx->gg.nnodes, DGAccumulator_sum, dg_avg, DGMax, local_infinity, ctx);
  // FP: "2 -> 3;
}
void SSSPSanityCheck_masterNodes_cuda(uint64_t & DGAccumulator_sum, uint64_t & dg_avg, uint32_t & DGMax, const uint32_t & local_infinity, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  SSSPSanityCheck_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, DGAccumulator_sum, dg_avg, DGMax, local_infinity, ctx);
  // FP: "2 -> 3;
}
void SSSPSanityCheck_nodesWithEdges_cuda(uint64_t & DGAccumulator_sum, uint64_t & dg_avg, uint32_t & DGMax, const uint32_t & local_infinity, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  SSSPSanityCheck_cuda(0, ctx->numNodesWithEdges, DGAccumulator_sum, dg_avg, DGMax, local_infinity, ctx);
  // FP: "2 -> 3;
}