#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"
#include "hipcub/hipcub.hpp"
#include "cub/util_allocator.cuh"
#include "thread_work.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=True $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ tb_lb=True $ hacks=set([]) $ np_factor=8 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=True $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
struct ThreadWork t_work;
bool enable_lb = true;
#include "kernels/reduce.cuh"
#include "pagerank_pull_cuda.cuh"
static const int __tb_PageRank = TB_SIZE;
static const int __tb_InitializeGraph = TB_SIZE;
__global__ void ResetGraph(CSRGraph graph, unsigned int __begin, unsigned int __end, const float  local_alpha, float * p_delta, uint32_t * p_nout, float * p_residual, float * p_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_value[src]    = 0;
      p_nout[src]     = 0;
      p_delta[src]    = 0;
      p_residual[src] = local_alpha;
    }
  }
  // FP: "10 -> 11;
}
__global__ void InitializeGraph_TB_LB(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_nout, DynamicBitset& bitset_nout, int * thread_prefix_work_wl, unsigned int num_items, PipeContextT<Worklist2> thread_src_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ unsigned int total_work;
  __shared__ unsigned block_start_src_index;
  __shared__ unsigned block_end_src_index;
  unsigned my_work;
  unsigned src;
  unsigned int offset;
  unsigned int current_work;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  unsigned blockdim_x = BLOCK_DIM_X;
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  // FP: "9 -> 10;
  total_work = thread_prefix_work_wl[num_items - 1];
  // FP: "10 -> 11;
  my_work = ceilf((float)(total_work) / (float) nthreads);
  // FP: "11 -> 12;

  // FP: "12 -> 13;
  __syncthreads();
  // FP: "13 -> 14;

  // FP: "14 -> 15;
  if (my_work != 0)
  {
    current_work = tid;
  }
  // FP: "17 -> 18;
  for (unsigned i =0; i < my_work; i++)
  {
    unsigned int block_start_work;
    unsigned int block_end_work;
    if (threadIdx.x == 0)
    {
      if (current_work < total_work)
      {
        block_start_work = current_work;
        block_end_work=current_work + blockdim_x - 1;
        if (block_end_work >= total_work)
        {
          block_end_work = total_work - 1;
        }
        block_start_src_index = compute_src_and_offset(0, num_items - 1,  block_start_work+1, thread_prefix_work_wl, num_items,offset);
        block_end_src_index = compute_src_and_offset(0, num_items - 1, block_end_work+1, thread_prefix_work_wl, num_items, offset);
      }
    }
    __syncthreads();

    if (current_work < total_work)
    {
      unsigned src_index;
      index_type nbr;
      src_index = compute_src_and_offset(block_start_src_index, block_end_src_index, current_work+1, thread_prefix_work_wl,num_items, offset);
      src= thread_src_wl.in_wl().dwl[src_index];
      nbr = (graph).getFirstEdge(src)+ offset;
      {
        index_type dst;
        dst = graph.getAbsDestination(nbr);
        atomicTestAdd(&p_nout[dst], (uint32_t)1);
        bitset_nout.set(dst);
      }
      current_work = current_work + nthreads;
    }
  }
  // FP: "43 -> 44;
}
__global__ void Inspect_InitializeGraph(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_nout, DynamicBitset& bitset_nout, PipeContextT<Worklist2> thread_work_wl, PipeContextT<Worklist2> thread_src_wl, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    int index;
    bool pop  = src < __end && ((( src < (graph).nnodes ) && ( (graph).getOutDegree(src) >= DEGREE_LIMIT)) ? true: false);
    if (pop)
    {
    }
    if (!pop)
    {
      continue;
    }
    if (pop)
    {
      index = thread_work_wl.in_wl().push_range(1) ;
      thread_src_wl.in_wl().push_range(1);
      thread_work_wl.in_wl().dwl[index] = (graph).getOutDegree(src);
      thread_src_wl.in_wl().dwl[index] = src;
    }
  }
  // FP: "13 -> 14;
}
__global__ void InitializeGraph(CSRGraph graph, unsigned int __begin, unsigned int __end, uint32_t * p_nout, DynamicBitset& bitset_nout, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_InitializeGraph;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  unsigned d_limit = DEGREE_LIMIT;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end && ((( src < (graph).nnodes ) && ( (graph).getOutDegree(src) < DEGREE_LIMIT)) ? true: false);
    // FP: "7 -> 8;
    if (pop)
    {
    }
    // FP: "9 -> 10;
    // FP: "12 -> 13;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "13 -> 14;
    __shared__ struct { ; } _np_closure [TB_SIZE];
    // FP: "14 -> 15;
    // FP: "15 -> 16;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "18 -> 19;
    // FP: "19 -> 20;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "20 -> 21;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "21 -> 22;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "24 -> 25;
    __syncthreads();
    // FP: "25 -> 26;
    while (true)
    {
      // FP: "26 -> 27;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "29 -> 30;
      __syncthreads();
      // FP: "30 -> 31;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "31 -> 32;
        __syncthreads();
        // FP: "32 -> 33;
        break;
      }
      // FP: "34 -> 35;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "37 -> 38;
      __syncthreads();
      // FP: "38 -> 39;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "39 -> 40;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "42 -> 43;
      assert(nps.tb.src < __kernel_tb_size);
      // FP: "43 -> 44;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicTestAdd(&p_nout[dst], (uint32_t)1);
          bitset_nout.set(dst);
        }
      }
      // FP: "51 -> 52;
      __syncthreads();
    }
    // FP: "53 -> 54;

    // FP: "54 -> 55;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "55 -> 56;
      const int _np_laneid = hipcub::LaneId();
      // FP: "56 -> 57;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(nbr);
            atomicTestAdd(&p_nout[dst], (uint32_t)1);
            bitset_nout.set(dst);
          }
        }
      }
      // FP: "74 -> 75;
      __syncthreads();
      // FP: "75 -> 76;
    }

    // FP: "76 -> 77;
    __syncthreads();
    // FP: "77 -> 78;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "78 -> 79;
    while (_np.work())
    {
      // FP: "79 -> 80;
      int _np_i =0;
      // FP: "80 -> 81;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "81 -> 82;
      __syncthreads();
      // FP: "82 -> 83;

      // FP: "83 -> 84;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          atomicTestAdd(&p_nout[dst], (uint32_t)1);
          bitset_nout.set(dst);
        }
      }
      // FP: "92 -> 93;
      _np.execute_round_done(ITSIZE);
      // FP: "93 -> 94;
      __syncthreads();
    }
    // FP: "95 -> 96;
    assert(threadIdx.x < __kernel_tb_size);
  }
  // FP: "97 -> 98;
}
__global__ void PageRank_delta(CSRGraph graph, unsigned int __begin, unsigned int __end, const float  local_alpha, float local_tolerance, float * p_delta, uint32_t * p_nout, float * p_residual, float * p_value, HGAccumulator<unsigned int> active_vertices)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<unsigned int, TB_SIZE>::TempStorage active_vertices_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  active_vertices.thread_entry();
  // FP: "3 -> 4;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      p_delta[src] = 0;
      if (p_residual[src] > 0)
      {
        p_value[src] += p_residual[src];
        if (p_residual[src] > local_tolerance)
        {
          if (p_nout[src] > 0)
          {
            p_delta[src] = p_residual[src] * (1 - local_alpha) / p_nout[src];
            active_vertices.reduce( 1);
          }
        }
        p_residual[src] = 0;
      }
    }
  }
  // FP: "19 -> 20;
  active_vertices.thread_exit<hipcub::BlockReduce<unsigned int, TB_SIZE> >(active_vertices_ts);
  // FP: "20 -> 21;
}
__global__ void PageRank_TB_LB(CSRGraph graph, unsigned int __begin, unsigned int __end, float * p_delta, float * p_residual, DynamicBitset& bitset_residual, int * thread_prefix_work_wl, unsigned int num_items, PipeContextT<Worklist2> thread_src_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ unsigned int total_work;
  __shared__ unsigned block_start_src_index;
  __shared__ unsigned block_end_src_index;
  unsigned my_work;
  unsigned src;
  unsigned int offset;
  unsigned int current_work;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  unsigned blockdim_x = BLOCK_DIM_X;
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  // FP: "9 -> 10;
  total_work = thread_prefix_work_wl[num_items - 1];
  // FP: "10 -> 11;
  my_work = ceilf((float)(total_work) / (float) nthreads);
  // FP: "11 -> 12;

  // FP: "12 -> 13;
  __syncthreads();
  // FP: "13 -> 14;

  // FP: "14 -> 15;
  if (my_work != 0)
  {
    current_work = tid;
  }
  // FP: "17 -> 18;
  for (unsigned i =0; i < my_work; i++)
  {
    unsigned int block_start_work;
    unsigned int block_end_work;
    if (threadIdx.x == 0)
    {
      if (current_work < total_work)
      {
        block_start_work = current_work;
        block_end_work=current_work + blockdim_x - 1;
        if (block_end_work >= total_work)
        {
          block_end_work = total_work - 1;
        }
        block_start_src_index = compute_src_and_offset(0, num_items - 1,  block_start_work+1, thread_prefix_work_wl, num_items,offset);
        block_end_src_index = compute_src_and_offset(0, num_items - 1, block_end_work+1, thread_prefix_work_wl, num_items, offset);
      }
    }
    __syncthreads();

    if (current_work < total_work)
    {
      unsigned src_index;
      index_type nbr;
      src_index = compute_src_and_offset(block_start_src_index, block_end_src_index, current_work+1, thread_prefix_work_wl,num_items, offset);
      src= thread_src_wl.in_wl().dwl[src_index];
      nbr = (graph).getFirstEdge(src)+ offset;
      {
        index_type dst;
        dst = graph.getAbsDestination(nbr);
        if (p_delta[dst] > 0)
        {
          atomicTestAdd(&p_residual[src], p_delta[dst]);
          bitset_residual.set(src);
        }
      }
      current_work = current_work + nthreads;
    }
  }
  // FP: "45 -> 46;
}
__global__ void Inspect_PageRank(CSRGraph graph, unsigned int __begin, unsigned int __end, float * p_delta, float * p_residual, DynamicBitset& bitset_residual, PipeContextT<Worklist2> thread_work_wl, PipeContextT<Worklist2> thread_src_wl, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  index_type src_end;
  // FP: "1 -> 2;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    int index;
    bool pop  = src < __end && ((( src < (graph).nnodes ) && ( (graph).getOutDegree(src) >= DEGREE_LIMIT)) ? true: false);
    if (pop)
    {
    }
    if (!pop)
    {
      continue;
    }
    if (pop)
    {
      index = thread_work_wl.in_wl().push_range(1) ;
      thread_src_wl.in_wl().push_range(1);
      thread_work_wl.in_wl().dwl[index] = (graph).getOutDegree(src);
      thread_src_wl.in_wl().dwl[index] = src;
    }
  }
  // FP: "13 -> 14;
}
__global__ void PageRank(CSRGraph graph, unsigned int __begin, unsigned int __end, float * p_delta, float * p_residual, DynamicBitset& bitset_residual, bool enable_lb)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = __tb_PageRank;
  index_type src_end;
  index_type src_rup;
  // FP: "1 -> 2;
  const int _NP_CROSSOVER_WP = 32;
  const int _NP_CROSSOVER_TB = __kernel_tb_size;
  // FP: "2 -> 3;
  const int BLKSIZE = __kernel_tb_size;
  const int ITSIZE = BLKSIZE * 8;
  unsigned d_limit = DEGREE_LIMIT;
  // FP: "3 -> 4;

  typedef hipcub::BlockScan<multiple_sum<2, index_type>, BLKSIZE> BlockScan;
  typedef union np_shared<BlockScan::TempStorage, index_type, struct tb_np, struct warp_np<__kernel_tb_size/32>, struct fg_np<ITSIZE> > npsTy;

  // FP: "4 -> 5;
  __shared__ npsTy nps ;
  // FP: "5 -> 6;
  src_end = __end;
  src_rup = ((__begin) + roundup(((__end) - (__begin)), (blockDim.x)));
  for (index_type src = __begin + tid; src < src_rup; src += nthreads)
  {
    multiple_sum<2, index_type> _np_mps;
    multiple_sum<2, index_type> _np_mps_total;
    // FP: "6 -> 7;
    bool pop  = src < __end && ((( src < (graph).nnodes ) && ( (graph).getOutDegree(src) < DEGREE_LIMIT)) ? true: false);
    // FP: "7 -> 8;
    if (pop)
    {
    }
    // FP: "9 -> 10;
    // FP: "12 -> 13;
    struct NPInspector1 _np = {0,0,0,0,0,0};
    // FP: "13 -> 14;
    __shared__ struct { index_type src; } _np_closure [TB_SIZE];
    // FP: "14 -> 15;
    _np_closure[threadIdx.x].src = src;
    // FP: "15 -> 16;
    if (pop)
    {
      _np.size = (graph).getOutDegree(src);
      _np.start = (graph).getFirstEdge(src);
    }
    // FP: "18 -> 19;
    // FP: "19 -> 20;
    _np_mps.el[0] = _np.size >= _NP_CROSSOVER_WP ? _np.size : 0;
    _np_mps.el[1] = _np.size < _NP_CROSSOVER_WP ? _np.size : 0;
    // FP: "20 -> 21;
    BlockScan(nps.temp_storage).ExclusiveSum(_np_mps, _np_mps, _np_mps_total);
    // FP: "21 -> 22;
    if (threadIdx.x == 0)
    {
      nps.tb.owner = MAX_TB_SIZE + 1;
    }
    // FP: "24 -> 25;
    __syncthreads();
    // FP: "25 -> 26;
    while (true)
    {
      // FP: "26 -> 27;
      if (_np.size >= _NP_CROSSOVER_TB)
      {
        nps.tb.owner = threadIdx.x;
      }
      // FP: "29 -> 30;
      __syncthreads();
      // FP: "30 -> 31;
      if (nps.tb.owner == MAX_TB_SIZE + 1)
      {
        // FP: "31 -> 32;
        __syncthreads();
        // FP: "32 -> 33;
        break;
      }
      // FP: "34 -> 35;
      if (nps.tb.owner == threadIdx.x)
      {
        nps.tb.start = _np.start;
        nps.tb.size = _np.size;
        nps.tb.src = threadIdx.x;
        _np.start = 0;
        _np.size = 0;
      }
      // FP: "37 -> 38;
      __syncthreads();
      // FP: "38 -> 39;
      int ns = nps.tb.start;
      int ne = nps.tb.size;
      // FP: "39 -> 40;
      if (nps.tb.src == threadIdx.x)
      {
        nps.tb.owner = MAX_TB_SIZE + 1;
      }
      // FP: "42 -> 43;
      assert(nps.tb.src < __kernel_tb_size);
      src = _np_closure[nps.tb.src].src;
      // FP: "43 -> 44;
      for (int _np_j = threadIdx.x; _np_j < ne; _np_j += BLKSIZE)
      {
        index_type nbr;
        nbr = ns +_np_j;
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          if (p_delta[dst] > 0)
          {
            atomicTestAdd(&p_residual[src], p_delta[dst]);
            bitset_residual.set(src);
          }
        }
      }
      // FP: "53 -> 54;
      __syncthreads();
    }
    // FP: "55 -> 56;

    // FP: "56 -> 57;
    {
      const int warpid = threadIdx.x / 32;
      // FP: "57 -> 58;
      const int _np_laneid = hipcub::LaneId();
      // FP: "58 -> 59;
      while (__any(_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB))
      {
        if (_np.size >= _NP_CROSSOVER_WP && _np.size < _NP_CROSSOVER_TB)
        {
          nps.warp.owner[warpid] = _np_laneid;
        }
        if (nps.warp.owner[warpid] == _np_laneid)
        {
          nps.warp.start[warpid] = _np.start;
          nps.warp.size[warpid] = _np.size;
          nps.warp.src[warpid] = threadIdx.x;
          _np.start = 0;
          _np.size = 0;
        }
        index_type _np_w_start = nps.warp.start[warpid];
        index_type _np_w_size = nps.warp.size[warpid];
        assert(nps.warp.src[warpid] < __kernel_tb_size);
        src = _np_closure[nps.warp.src[warpid]].src;
        for (int _np_ii = _np_laneid; _np_ii < _np_w_size; _np_ii += 32)
        {
          index_type nbr;
          nbr = _np_w_start +_np_ii;
          {
            index_type dst;
            dst = graph.getAbsDestination(nbr);
            if (p_delta[dst] > 0)
            {
              atomicTestAdd(&p_residual[src], p_delta[dst]);
              bitset_residual.set(src);
            }
          }
        }
      }
      // FP: "78 -> 79;
      __syncthreads();
      // FP: "79 -> 80;
    }

    // FP: "80 -> 81;
    __syncthreads();
    // FP: "81 -> 82;
    _np.total = _np_mps_total.el[1];
    _np.offset = _np_mps.el[1];
    // FP: "82 -> 83;
    while (_np.work())
    {
      // FP: "83 -> 84;
      int _np_i =0;
      // FP: "84 -> 85;
      _np.inspect2(nps.fg.itvalue, nps.fg.src, ITSIZE, threadIdx.x);
      // FP: "85 -> 86;
      __syncthreads();
      // FP: "86 -> 87;

      // FP: "87 -> 88;
      for (_np_i = threadIdx.x; _np_i < ITSIZE && _np.valid(_np_i); _np_i += BLKSIZE)
      {
        index_type nbr;
        assert(nps.fg.src[_np_i] < __kernel_tb_size);
        src = _np_closure[nps.fg.src[_np_i]].src;
        nbr= nps.fg.itvalue[_np_i];
        {
          index_type dst;
          dst = graph.getAbsDestination(nbr);
          if (p_delta[dst] > 0)
          {
            atomicTestAdd(&p_residual[src], p_delta[dst]);
            bitset_residual.set(src);
          }
        }
      }
      // FP: "98 -> 99;
      _np.execute_round_done(ITSIZE);
      // FP: "99 -> 100;
      __syncthreads();
    }
    // FP: "101 -> 102;
    assert(threadIdx.x < __kernel_tb_size);
    src = _np_closure[threadIdx.x].src;
  }
  // FP: "103 -> 104;
}
__global__ void PageRankSanity(CSRGraph graph, unsigned int __begin, unsigned int __end, float local_tolerance, float * p_residual, float * p_value, HGAccumulator<uint64_t> DGAccumulator_residual_over_tolerance, HGAccumulator<float> DGAccumulator_sum, HGAccumulator<float> DGAccumulator_sum_residual, HGReduceMax<float> max_residual, HGReduceMax<float> max_value, HGReduceMin<float> min_residual, HGReduceMin<float> min_value)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  __shared__ hipcub::BlockReduce<uint64_t, TB_SIZE>::TempStorage DGAccumulator_residual_over_tolerance_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage DGAccumulator_sum_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage DGAccumulator_sum_residual_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage max_residual_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage max_value_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage min_residual_ts;
  __shared__ hipcub::BlockReduce<float, TB_SIZE>::TempStorage min_value_ts;
  index_type src_end;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  DGAccumulator_residual_over_tolerance.thread_entry();
  // FP: "3 -> 4;
  // FP: "4 -> 5;
  DGAccumulator_sum.thread_entry();
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  DGAccumulator_sum_residual.thread_entry();
  // FP: "7 -> 8;
  // FP: "8 -> 9;
  max_residual.thread_entry();
  // FP: "9 -> 10;
  // FP: "10 -> 11;
  max_value.thread_entry();
  // FP: "11 -> 12;
  // FP: "12 -> 13;
  min_residual.thread_entry();
  // FP: "13 -> 14;
  // FP: "14 -> 15;
  min_value.thread_entry();
  // FP: "15 -> 16;
  src_end = __end;
  for (index_type src = __begin + tid; src < src_end; src += nthreads)
  {
    bool pop  = src < __end;
    if (pop)
    {
      max_value.reduce(p_value[src]);
      min_value.reduce(p_value[src]);
      max_residual.reduce(p_residual[src]);
      min_residual.reduce(p_residual[src]);
      DGAccumulator_sum.reduce( p_value[src]);
      DGAccumulator_sum.reduce( p_residual[src]);
      if (p_residual[src] > local_tolerance)
      {
        DGAccumulator_residual_over_tolerance.reduce( 1);
      }
    }
  }
  // FP: "29 -> 30;
  DGAccumulator_residual_over_tolerance.thread_exit<hipcub::BlockReduce<uint64_t, TB_SIZE> >(DGAccumulator_residual_over_tolerance_ts);
  // FP: "30 -> 31;
  DGAccumulator_sum.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(DGAccumulator_sum_ts);
  // FP: "31 -> 32;
  DGAccumulator_sum_residual.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(DGAccumulator_sum_residual_ts);
  // FP: "32 -> 33;
  max_residual.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(max_residual_ts);
  // FP: "33 -> 34;
  max_value.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(max_value_ts);
  // FP: "34 -> 35;
  min_residual.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(min_residual_ts);
  // FP: "35 -> 36;
  min_value.thread_exit<hipcub::BlockReduce<float, TB_SIZE> >(min_value_ts);
  // FP: "36 -> 37;
}
void ResetGraph_cuda(unsigned int  __begin, unsigned int  __end, const float & local_alpha, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  ResetGraph <<<blocks, threads>>>(ctx->gg, __begin, __end, local_alpha, ctx->delta.data.gpu_wr_ptr(), ctx->nout.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr());
  hipDeviceSynchronize();
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void ResetGraph_allNodes_cuda(const float & local_alpha, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(0, ctx->gg.nnodes, local_alpha, ctx);
  // FP: "2 -> 3;
}
void ResetGraph_masterNodes_cuda(const float & local_alpha, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, local_alpha, ctx);
  // FP: "2 -> 3;
}
void ResetGraph_nodesWithEdges_cuda(const float & local_alpha, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  ResetGraph_cuda(0, ctx->numNodesWithEdges, local_alpha, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  t_work.init_thread_work(ctx->gg.nnodes);
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  if (enable_lb)
  {
    t_work.reset_thread_work();
    Inspect_InitializeGraph <<<blocks, __tb_InitializeGraph>>>(ctx->gg, __begin, __end, ctx->nout.data.gpu_wr_ptr(), *(ctx->nout.is_updated.gpu_rd_ptr()), t_work.thread_work_wl, t_work.thread_src_wl, enable_lb);
    hipDeviceSynchronize();
    int num_items = t_work.thread_work_wl.in_wl().nitems();
    if (num_items != 0)
    {
      t_work.compute_prefix_sum();
      hipDeviceSynchronize();
      InitializeGraph_TB_LB <<<blocks, __tb_InitializeGraph>>>(ctx->gg, __begin, __end, ctx->nout.data.gpu_wr_ptr(), *(ctx->nout.is_updated.gpu_rd_ptr()), t_work.thread_prefix_work_wl.gpu_wr_ptr(), num_items, t_work.thread_src_wl);
      hipDeviceSynchronize();
    }
  }
  InitializeGraph <<<blocks, __tb_InitializeGraph>>>(ctx->gg, __begin, __end, ctx->nout.data.gpu_wr_ptr(), *(ctx->nout.is_updated.gpu_rd_ptr()), enable_lb);
  hipDeviceSynchronize();
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void InitializeGraph_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void InitializeGraph_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  InitializeGraph_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void PageRank_delta_cuda(unsigned int  __begin, unsigned int  __end, unsigned int & active_vertices, const float & local_alpha, float local_tolerance, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<unsigned int> _active_vertices;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<unsigned int> active_verticesval  = Shared<unsigned int>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(active_verticesval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _active_vertices.rv = active_verticesval.gpu_wr_ptr();
  // FP: "8 -> 9;
  PageRank_delta <<<blocks, threads>>>(ctx->gg, __begin, __end, local_alpha, local_tolerance, ctx->delta.data.gpu_wr_ptr(), ctx->nout.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr(), _active_vertices);
  hipDeviceSynchronize();
  // FP: "9 -> 10;
  check_cuda_kernel;
  // FP: "10 -> 11;
  active_vertices = *(active_verticesval.cpu_rd_ptr());
  // FP: "11 -> 12;
}
void PageRank_delta_allNodes_cuda(unsigned int & active_vertices, const float & local_alpha, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_delta_cuda(0, ctx->gg.nnodes, active_vertices, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRank_delta_masterNodes_cuda(unsigned int & active_vertices, const float & local_alpha, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_delta_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, active_vertices, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRank_delta_nodesWithEdges_cuda(unsigned int & active_vertices, const float & local_alpha, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_delta_cuda(0, ctx->numNodesWithEdges, active_vertices, local_alpha, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRank_cuda(unsigned int  __begin, unsigned int  __end, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  if (enable_lb)
  {
    t_work.reset_thread_work();
    Inspect_PageRank <<<blocks, __tb_PageRank>>>(ctx->gg, __begin, __end, ctx->delta.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), *(ctx->residual.is_updated.gpu_rd_ptr()), t_work.thread_work_wl, t_work.thread_src_wl, enable_lb);
    hipDeviceSynchronize();
    int num_items = t_work.thread_work_wl.in_wl().nitems();
    if (num_items != 0)
    {
      t_work.compute_prefix_sum();
      hipDeviceSynchronize();
      PageRank_TB_LB <<<blocks, __tb_PageRank>>>(ctx->gg, __begin, __end, ctx->delta.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), *(ctx->residual.is_updated.gpu_rd_ptr()), t_work.thread_prefix_work_wl.gpu_wr_ptr(), num_items, t_work.thread_src_wl);
      hipDeviceSynchronize();
    }
  }
  PageRank <<<blocks, __tb_PageRank>>>(ctx->gg, __begin, __end, ctx->delta.data.gpu_wr_ptr(), ctx->residual.data.gpu_wr_ptr(), *(ctx->residual.is_updated.gpu_rd_ptr()), enable_lb);
  hipDeviceSynchronize();
  // FP: "5 -> 6;
  check_cuda_kernel;
  // FP: "6 -> 7;
}
void PageRank_allNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(0, ctx->gg.nnodes, ctx);
  // FP: "2 -> 3;
}
void PageRank_masterNodes_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, ctx);
  // FP: "2 -> 3;
}
void PageRank_nodesWithEdges_cuda(struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRank_cuda(0, ctx->numNodesWithEdges, ctx);
  // FP: "2 -> 3;
}
void PageRankSanity_cuda(unsigned int  __begin, unsigned int  __end, uint64_t & DGAccumulator_residual_over_tolerance, float & DGAccumulator_sum, float & DGAccumulator_sum_residual, float & max_residual, float & max_value, float & min_residual, float & min_value, float local_tolerance, struct CUDA_Context*  ctx)
{
  dim3 blocks;
  dim3 threads;
  HGAccumulator<uint64_t> _DGAccumulator_residual_over_tolerance;
  HGAccumulator<float> _DGAccumulator_sum;
  HGAccumulator<float> _DGAccumulator_sum_residual;
  HGReduceMax<float> _max_residual;
  HGReduceMax<float> _max_value;
  HGReduceMin<float> _min_residual;
  HGReduceMin<float> _min_value;
  // FP: "1 -> 2;
  // FP: "2 -> 3;
  // FP: "3 -> 4;
  kernel_sizing(blocks, threads);
  // FP: "4 -> 5;
  Shared<uint64_t> DGAccumulator_residual_over_toleranceval  = Shared<uint64_t>(1);
  // FP: "5 -> 6;
  // FP: "6 -> 7;
  *(DGAccumulator_residual_over_toleranceval.cpu_wr_ptr()) = 0;
  // FP: "7 -> 8;
  _DGAccumulator_residual_over_tolerance.rv = DGAccumulator_residual_over_toleranceval.gpu_wr_ptr();
  // FP: "8 -> 9;
  Shared<float> DGAccumulator_sumval  = Shared<float>(1);
  // FP: "9 -> 10;
  // FP: "10 -> 11;
  *(DGAccumulator_sumval.cpu_wr_ptr()) = 0;
  // FP: "11 -> 12;
  _DGAccumulator_sum.rv = DGAccumulator_sumval.gpu_wr_ptr();
  // FP: "12 -> 13;
  Shared<float> DGAccumulator_sum_residualval  = Shared<float>(1);
  // FP: "13 -> 14;
  // FP: "14 -> 15;
  *(DGAccumulator_sum_residualval.cpu_wr_ptr()) = 0;
  // FP: "15 -> 16;
  _DGAccumulator_sum_residual.rv = DGAccumulator_sum_residualval.gpu_wr_ptr();
  // FP: "16 -> 17;
  Shared<float> max_residualval  = Shared<float>(1);
  // FP: "17 -> 18;
  // FP: "18 -> 19;
  *(max_residualval.cpu_wr_ptr()) = 0;
  // FP: "19 -> 20;
  _max_residual.rv = max_residualval.gpu_wr_ptr();
  // FP: "20 -> 21;
  Shared<float> max_valueval  = Shared<float>(1);
  // FP: "21 -> 22;
  // FP: "22 -> 23;
  *(max_valueval.cpu_wr_ptr()) = 0;
  // FP: "23 -> 24;
  _max_value.rv = max_valueval.gpu_wr_ptr();
  // FP: "24 -> 25;
  Shared<float> min_residualval  = Shared<float>(1);
  // FP: "25 -> 26;
  // FP: "26 -> 27;
  *(min_residualval.cpu_wr_ptr()) = 0;
  // FP: "27 -> 28;
  _min_residual.rv = min_residualval.gpu_wr_ptr();
  // FP: "28 -> 29;
  Shared<float> min_valueval  = Shared<float>(1);
  // FP: "29 -> 30;
  // FP: "30 -> 31;
  *(min_valueval.cpu_wr_ptr()) = 0;
  // FP: "31 -> 32;
  _min_value.rv = min_valueval.gpu_wr_ptr();
  // FP: "32 -> 33;
  PageRankSanity <<<blocks, threads>>>(ctx->gg, __begin, __end, local_tolerance, ctx->residual.data.gpu_wr_ptr(), ctx->value.data.gpu_wr_ptr(), _DGAccumulator_residual_over_tolerance, _DGAccumulator_sum, _DGAccumulator_sum_residual, _max_residual, _max_value, _min_residual, _min_value);
  hipDeviceSynchronize();
  // FP: "33 -> 34;
  check_cuda_kernel;
  // FP: "34 -> 35;
  DGAccumulator_residual_over_tolerance = *(DGAccumulator_residual_over_toleranceval.cpu_rd_ptr());
  // FP: "35 -> 36;
  DGAccumulator_sum = *(DGAccumulator_sumval.cpu_rd_ptr());
  // FP: "36 -> 37;
  DGAccumulator_sum_residual = *(DGAccumulator_sum_residualval.cpu_rd_ptr());
  // FP: "37 -> 38;
  max_residual = *(max_residualval.cpu_rd_ptr());
  // FP: "38 -> 39;
  max_value = *(max_valueval.cpu_rd_ptr());
  // FP: "39 -> 40;
  min_residual = *(min_residualval.cpu_rd_ptr());
  // FP: "40 -> 41;
  min_value = *(min_valueval.cpu_rd_ptr());
  // FP: "41 -> 42;
}
void PageRankSanity_allNodes_cuda(uint64_t & DGAccumulator_residual_over_tolerance, float & DGAccumulator_sum, float & DGAccumulator_sum_residual, float & max_residual, float & max_value, float & min_residual, float & min_value, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRankSanity_cuda(0, ctx->gg.nnodes, DGAccumulator_residual_over_tolerance, DGAccumulator_sum, DGAccumulator_sum_residual, max_residual, max_value, min_residual, min_value, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRankSanity_masterNodes_cuda(uint64_t & DGAccumulator_residual_over_tolerance, float & DGAccumulator_sum, float & DGAccumulator_sum_residual, float & max_residual, float & max_value, float & min_residual, float & min_value, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRankSanity_cuda(ctx->beginMaster, ctx->beginMaster + ctx->numOwned, DGAccumulator_residual_over_tolerance, DGAccumulator_sum, DGAccumulator_sum_residual, max_residual, max_value, min_residual, min_value, local_tolerance, ctx);
  // FP: "2 -> 3;
}
void PageRankSanity_nodesWithEdges_cuda(uint64_t & DGAccumulator_residual_over_tolerance, float & DGAccumulator_sum, float & DGAccumulator_sum_residual, float & max_residual, float & max_value, float & min_residual, float & min_value, float local_tolerance, struct CUDA_Context*  ctx)
{
  // FP: "1 -> 2;
  PageRankSanity_cuda(0, ctx->numNodesWithEdges, DGAccumulator_residual_over_tolerance, DGAccumulator_sum, DGAccumulator_sum_residual, max_residual, max_value, min_residual, min_value, local_tolerance, ctx);
  // FP: "2 -> 3;
}